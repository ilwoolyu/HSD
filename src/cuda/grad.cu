#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipsparse.h>
#include "grad.h"
#include "geom.h"
#include "geom.cu"

__global__ void DA_kernel(double *A, double *B, double *C, int num_rows, int num_cols)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < num_rows)
		for (int col = 0; col < num_cols; col++)
			C[row * num_cols + col] = A[row] * B[row * num_cols + col];
}

__global__ void SA_kernel(double scalar, double *A, int num_rows, int num_cols)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < num_rows)
		for (int col = 0; col < num_cols; col++)
			A[row * num_cols + col] *= scalar;
}

__global__ void PA_kernel(double *A, double *B, int num_rows, int num_cols)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < num_rows)
		for (int col = 0; col < num_cols; col++)
			A[row * num_cols + col] += B[row * num_cols + col];
}

__global__ void gradient_properties_kernel(const float *vertex, int nVertex, const int *face, int nFace, const float *feature, const float *propertySamples, int nSamples, const double *variance, const float *property, const float *pole, const double *Y, const double *coeff, int degree, int deg_beg, int deg_end, double normalization, const double *m_bar, const float *u1, const float *u2, const int *fid, double *gradient, double *gradient_raw, double *gradient_diag, double *dEdx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < nSamples)
	{
		cuCoordinate coord;
		int nCoeff = (degree + 1) * (degree + 1);
		int n = (deg_end + 1) * (deg_end + 1);
		int n0 = deg_beg * deg_beg;
		int size = n - n0;

		// spherical coordinate
		const float *y = &propertySamples[i * 3];
		float cY[3];
		int id1 = face[fid[i] * 3 + 0];
		int id2 = face[fid[i] * 3 + 1];
		int id3 = face[fid[i] * 3 + 2];

		// approximation of Y
		const float *v1 = &vertex[id1 * 3];
		const float *v2 = &vertex[id2 * 3];
		const float *v3 = &vertex[id3 * 3];
		
		cuVector N = cuVector(v1, v2).cross(cuVector(v2, v3));
		float area = N.norm();
		N.unit();
		cuVector Yproj = cuVector(y) * ((cuVector(v1) * N) / (cuVector(y) * N));
		coord.cart2bary((float *)v1, (float *)v2, (float *)v3, (float *)Yproj.fv(), cY);

		const double *Y1 = &Y[nCoeff * id1];
		const double *Y2 = &Y[nCoeff * id2];
		const double *Y3 = &Y[nCoeff * id3];

		double delta[3] = {0, 0, 0};
		for (int j = 0; j < nCoeff; j++)
		{
			double Y = (Y1[j] * cY[0] + Y2[j] * cY[1] + Y3[j] * cY[2]);
			delta[0] += Y * coeff[j];
			delta[1] += Y * coeff[nCoeff + j];
			delta[2] += Y * coeff[nCoeff * 2 + j];
		}
		// m
		double m = feature[i];

		// z
		float z_hat[3];
		coord.sph2cart(pole[0], pole[1], z_hat);
		float z_dot_orth[3]; const float *z_dot_orth_ = (cuVector(u1) * (float)delta[1] + cuVector(u2) * (float)delta[2]).unit().fv();
		for (int j = 0; j < 3; j++) z_dot_orth[j] = z_dot_orth_[j];
		float degree = (float)sqrt(delta[1] * delta[1] + delta[2] * delta[2]);
		cuVector Z_ddot = cuVector(z_hat).cross(cuVector(z_dot_orth));
		float z_ddot[3]; const float *z_ddot_ = Z_ddot.unit().fv();
		for (int j = 0; j < 3; j++) z_ddot[j] = z_ddot_[j];
		float rot[9];
		coord.rotation(z_ddot, degree, rot);
		float z_dot[3];
		coord.rotPoint(z_hat, rot, z_dot);

		// dp/dx
		float nf[3]; const float *nf_ = N.fv();
		for (int j = 0; j < 3; j++) nf[j] = nf_[j];
		cuVector V1(v1), Yv(y);
		float v1N = V1 * N;
		float yN = Yv * N;
		double r1 = v1N / yN;
		double r2 = r1 / yN;
		double dpdx[3][3] = {{r1 - r2 * y[0] * nf[0], -r2 * y[0] * nf[1], -r2 * y[0] * nf[2]},
							{-r2 * y[1] * nf[0], r1 - r2 * y[1] * nf[1], -r2 * y[1] * nf[2]},
							{-r2 * y[2] * nf[0], -r2 * y[2] * nf[1], r1 - r2 * y[2] * nf[2]}};
		// dm/dp
		cuVector YP1(Yproj.fv(), v1), YP2(Yproj.fv(), v2), YP3(Yproj.fv(), v3);
		float m1 = property[id1];
		float m2 = property[id2];
		float m3 = property[id3];

		cuVector V1V2(v1, v2), V2V3(v2, v3), V3V1(v3, v1);
		float v1v2_norm = V1V2.norm();
		float v2v3_norm = V2V3.norm();
		float v3v1_norm = V3V1.norm();
		cuVector DA1DP = V2V3.cross(N).unit() * v2v3_norm * m1;
		cuVector DA2DP = V3V1.cross(N).unit() * v3v1_norm * m2;
		cuVector DA3DP = V1V2.cross(N).unit() * v1v2_norm * m3;
		cuVector DMDP = (DA1DP + DA2DP + DA3DP) / area;
		//cuVector DMDP = (DA1DP + DA2DP + DA3DP);	// canceled out: area
	
		const float *dmdp = DMDP.fv();
	
		// grad_m
		double grad_m[3] = {dpdx[0][0] * dmdp[0] + dpdx[0][1] * dmdp[1] + dpdx[0][2] * dmdp[2],
							dpdx[1][0] * dmdp[0] + dpdx[1][1] * dmdp[1] + dpdx[1][2] * dmdp[2],
							dpdx[2][0] * dmdp[0] + dpdx[2][1] * dmdp[1] + dpdx[2][2] * dmdp[2]};

		//cout << "grad_m: " << grad_m[0] << " " << grad_m[1] << " " << grad_m[2] << endl;
	
		// z_hatXu
		float z_hatXu1[3]; const float *z_hatXu1_ = cuVector(z_hat).cross(cuVector(u1)).fv();
		for (int j = 0; j < 3; j++) z_hatXu1[j] = z_hatXu1_[j];
		float z_hatXu2[3]; const float *z_hatXu2_ = cuVector(z_hat).cross(cuVector(u2)).fv();
		for (int j = 0; j < 3; j++) z_hatXu2[j] = z_hatXu2_[j];

		// [z_dot]x
		double z_dot_x[3][3] = {{0, -z_dot[2], z_dot[1]},
							 {z_dot[2], 0, -z_dot[0]},
							 {-z_dot[1], z_dot[0], 0}};
		// [z_hatXu1]x
		double z_hatXu1_x[3][3] = {{0, -z_hatXu1[2], z_hatXu1[1]},
								 {z_hatXu1[2], 0, -z_hatXu1[0]},
								 {-z_hatXu1[1], z_hatXu1[0], 0}};
		// [z_hatXu2]x
		double z_hatXu2_x[3][3] = {{0, -z_hatXu2[2], z_hatXu2[1]},
								 {z_hatXu2[2], 0, -z_hatXu2[0]},
								 {-z_hatXu2[1], z_hatXu2[0], 0}};

		// dx_hat/dgamma = (z_dot_x * x_hat) * y
		double dxdg = (z_dot_x[0][0] * y[0] + z_dot_x[0][1] * y[1] + z_dot_x[0][2] * y[2]) * grad_m[0] +
					(z_dot_x[1][0] * y[0] + z_dot_x[1][1] * y[1] + z_dot_x[1][2] * y[2]) * grad_m[1] +
					(z_dot_x[2][0] * y[0] + z_dot_x[2][1] * y[1] + z_dot_x[2][2] * y[2]) * grad_m[2];

		// dy/du1 = (z_hatXu1_x * y) * y
		double dxdu1 = (z_hatXu1_x[0][0] * y[0] + z_hatXu1_x[0][1] * y[1] + z_hatXu1_x[0][2] * y[2]) * grad_m[0] +
					(z_hatXu1_x[1][0] * y[0] + z_hatXu1_x[1][1] * y[1] + z_hatXu1_x[1][2] * y[2]) * grad_m[1] +
					(z_hatXu1_x[2][0] * y[0] + z_hatXu1_x[2][1] * y[1] + z_hatXu1_x[2][2] * y[2]) * grad_m[2];

		// dy/du2 = (z_hatXu1_x * y) * y
		double dxdu2 = (z_hatXu2_x[0][0] * y[0] + z_hatXu2_x[0][1] * y[1] + z_hatXu2_x[0][2] * y[2]) * grad_m[0] +
					(z_hatXu2_x[1][0] * y[0] + z_hatXu2_x[1][1] * y[1] + z_hatXu2_x[1][2] * y[2]) * grad_m[1] +
					(z_hatXu2_x[2][0] * y[0] + z_hatXu2_x[2][1] * y[1] + z_hatXu2_x[2][2] * y[2]) * grad_m[2];
		// dE/dx
		dEdx[i] = 2 * (m - m_bar[i]) / variance[i];
		//cout << "dxdg: " << dxdg << " dxdu1: " << dxdu1 << " dxdu2: " << dxdu2 << endl;
	
		gradient_diag[i] = normalization / variance[i];
		//totalArea += area;
		
		for (int j = n0; j < n; j++)
		{
			double nY = (Y1[j] * cY[0] + Y2[j] * cY[1] + Y3[j] * cY[2]);
			/*atomicAdd(&gradient[j], nY * dxdg * dEdx[i]);
			atomicAdd(&gradient[nCoeff + j], nY * dxdu1 * dEdx[i]);
			atomicAdd(&gradient[nCoeff * 2 + j], nY * dxdu2 * dEdx[i]);*/
			gradient_raw[size * 3 * i + j - n0] = nY * dxdg;
			gradient_raw[size * 3 * i + size + j - n0] = nY * dxdu1;
			gradient_raw[size * 3 * i + size * 2 + j - n0] = nY * dxdu2;
		}
	}
}

__global__ void dEdx_kernel(int nVertex, int degree, int deg_beg, int deg_end, double *gradient, double *gradient_raw, double *dEdx)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int n = (deg_end + 1) * (deg_end + 1);
	int n0 = deg_beg * deg_beg;
	int size = n - n0;

	if (x >= 0 && x < size * 3)
	{
		int k = x / size;
		int j = x % size + n0;
		int nCoeff = (degree + 1) * (degree + 1);
		gradient[nCoeff * k + j] = 0;
		for (int i = 0; i < nVertex; i++)
			gradient[nCoeff * k + j] += gradient_raw[size * 3 * i + size * k + j - n0] * dEdx[i];
	}
}

void Gradient::_ATB(double *d_A, int nr_rows_A, int nr_cols_A, double *d_B, int nr_cols_B, double *d_C)
{
	int m = nr_cols_A, n = nr_cols_B, k = nr_rows_A;
	int lda = m,ldb = n,ldc = m;
	const double alpha = 1;
	const double beta = 0;
	
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);
	
	hipblasDestroy(handle);
}

void Gradient::_DA(double *d_D, double *d_A, int nr_rows, int nr_cols, double *d_B)
{
	int blocksize = 256; // or any size up to 512
	int nblocks = (nr_rows + blocksize - 1) / blocksize;
	
	DA_kernel<<<nblocks,blocksize>>>(d_D, d_A, d_B, nr_rows, nr_cols);
}

void Gradient::_SA(double scalar, double *d_A, int nr_rows, int nr_cols)
{
	int blocksize = 256; // or any size up to 512
	int nblocks = (nr_rows + blocksize - 1) / blocksize;
	
	SA_kernel<<<nblocks,blocksize>>>(scalar, d_A, nr_rows, nr_cols);
}

void Gradient::_PA(double *d_A, double *d_B, int nr_rows, int nr_cols)
{
	int blocksize = 256; // or any size up to 512
	int nblocks = (nr_rows + blocksize - 1) / blocksize;
	
	PA_kernel<<<nblocks,blocksize>>>(d_A, d_B, nr_rows, nr_cols);
}

void Gradient::_ATDA(double *d_A, double *d_D, int nr_rows_A, int nr_cols_A, double *d_B)
{
	double *d_C;
	hipMalloc(&d_C, nr_rows_A * nr_cols_A * sizeof(double));
	_DA(d_D, d_A, nr_rows_A, nr_cols_A, d_C);
	_ATB(d_A, nr_rows_A, nr_cols_A, d_C, nr_cols_A, d_B);
	hipFree(d_C);
}

void Gradient::ATDA(double *h_A, double *h_D, int nr_rows_A, int nr_cols_A, double *h_B)
{
	double *d_A;
	double *d_D;
	double *d_B;
	double *d_C;
	hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(double));
	hipMalloc(&d_D, nr_rows_A * sizeof(double));
	hipMalloc(&d_B, nr_cols_A * nr_cols_A * sizeof(double));
	hipMalloc(&d_C, nr_rows_A * nr_cols_A * sizeof(double));
	hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_D, h_D, nr_rows_A * sizeof(double), hipMemcpyHostToDevice);

	_DA(d_D, d_A, nr_rows_A, nr_cols_A, d_C);
	_ATB(d_A, nr_rows_A, nr_cols_A, d_C, nr_cols_A, d_B);

	hipMemcpy(h_B, d_B, nr_cols_A * nr_cols_A * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_D);
	hipFree(d_B);
	hipFree(d_C);
}

void Gradient::ATB(double *h_A, int nr_rows_A, int nr_cols_A, double *h_B, int nr_cols_B, double *h_C)
{
	double *d_A;
	double *d_B;
	double *d_C;
	hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(double));
	hipMalloc(&d_B, nr_cols_A * nr_cols_B * sizeof(double));
	hipMalloc(&d_C, nr_rows_A * nr_cols_B * sizeof(double));
	hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nr_cols_A * nr_cols_B * sizeof(double), hipMemcpyHostToDevice);

	_ATB(d_A, nr_rows_A, nr_cols_A, d_B, nr_cols_B, d_C);

	hipMemcpy(h_C, d_C, nr_rows_A * nr_cols_B * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void Gradient::updateGradientProperties(const float *vertex, int nVertex, const int *face, int nFace, const float *feature, const float *propertySamples, int nSamples, const double *variance, const float *property, const float *pole, const double *Y, const double *coeff, int degree, int deg_beg, int deg_end, double normalization, const double *m_bar, const float *u1, const float *u2, const int *fid, double *gradient, double *M, bool hessian)
{
	int blocksize = 256; // or any size up to 512
	int nblocks = (nSamples + blocksize - 1) / blocksize;
	
	float *d_vertex;
	int *d_face;
	float *d_feature;
	double *d_variance;
	float *d_property;
	float *d_pole;
	float *d_propertySamples;
	double *d_Y;
	double *d_coeff;
	float *d_u1;
	float *d_u2;
	double *d_gradient;
	double *d_gradient_new;
	double *d_gradient_raw;
	double *d_gradient_diag;
	double *d_dEdx;
	double *d_m_bar;
	int *d_fid;
	double *d_M;
	double *d_M_new;
	int n = (deg_end + 1) * (deg_end + 1);
	int n0 = deg_beg * deg_beg;
	int size = n - n0;
	int nblocks2 = (size * 3 + blocksize - 1) / blocksize;
	
	hipMalloc(&d_vertex, nVertex * 3 * sizeof(float));
	hipMalloc(&d_face, nFace * 3 * sizeof(int));
	hipMalloc(&d_feature, nSamples * sizeof(float));
	hipMalloc(&d_variance, nSamples * sizeof(double));
	hipMalloc(&d_property, nVertex * sizeof(float));
	hipMalloc(&d_pole, 2 * sizeof(float));
	hipMalloc(&d_propertySamples, nSamples * 3 * sizeof(float));
	hipMalloc(&d_Y, nVertex * (degree + 1) * (degree + 1) * sizeof(double));
	hipMalloc(&d_coeff, (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_m_bar, nSamples * sizeof(double));
	hipMalloc(&d_fid, nSamples * sizeof(int));
	hipMalloc(&d_u1, 3 * sizeof(float));
	hipMalloc(&d_u2, 3 * sizeof(float));
	hipMalloc(&d_gradient, (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_gradient_new, (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_gradient_raw, nSamples * (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_gradient_diag, nSamples * sizeof(double));
	hipMalloc(&d_dEdx, nSamples * sizeof(double));
	hipMalloc(&d_M, size * 3 * size * 3 * sizeof(double));
	hipMalloc(&d_M_new, size * 3 * size * 3 * sizeof(double));
	
	hipMemcpy(d_vertex, vertex, nVertex * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_face, face, nFace * 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_feature, feature, nSamples * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_variance, variance, nSamples * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_property, property, nVertex * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_pole, pole, 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_propertySamples, propertySamples, nSamples * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, nVertex * (degree + 1) * (degree + 1) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_coeff, coeff, (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m_bar, m_bar, nSamples * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_fid, fid, nSamples * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_u1, u1, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u2, u2, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gradient, gradient, (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_M, M, size * 3 * size * 3 * sizeof(double), hipMemcpyHostToDevice);

	gradient_properties_kernel<<<nblocks,blocksize>>>(d_vertex, nVertex, d_face, nFace, d_feature, d_propertySamples, nSamples, d_variance, d_property, d_pole, d_Y, d_coeff, degree, deg_beg, deg_end, normalization, d_m_bar, d_u1, d_u2, d_fid, d_gradient_new, d_gradient_raw, d_gradient_diag, d_dEdx);
	dEdx_kernel<<<nblocks2,blocksize>>>(nSamples, degree, deg_beg, deg_end, d_gradient_new, d_gradient_raw, d_dEdx);
	_SA(normalization, d_gradient_new, (degree + 1) * (degree + 1) * 3, 1);
	_PA(d_gradient, d_gradient_new, (degree + 1) * (degree + 1) * 3, 1);
	hipMemcpy(gradient, d_gradient, (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyDeviceToHost);
	
	if (hessian)
	{
		_ATDA(d_gradient_raw, d_gradient_diag, nSamples, size * 3, d_M_new);
		_PA(d_M, d_M_new, size * 3, size * 3);
		hipMemcpy(M, d_M, size * 3 * size * 3 * sizeof(double), hipMemcpyDeviceToHost);
	}
	//hipMemcpy(gradient_raw, d_gradient_raw, nSamples * (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(gradient_diag, d_gradient_diag, nSamples * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_vertex);
	hipFree(d_feature);
	hipFree(d_variance);
	hipFree(d_face);
	hipFree(d_property);
	hipFree(d_pole);
	hipFree(d_propertySamples);
	hipFree(d_Y);
	hipFree(d_coeff);
	hipFree(d_u1);
	hipFree(d_u2);
	hipFree(d_m_bar);
	hipFree(d_fid);
	hipFree(d_gradient);
	hipFree(d_gradient_new);
	hipFree(d_gradient_raw);
	hipFree(d_gradient_diag);
	hipFree(d_dEdx);
	hipFree(d_M);
	hipFree(d_M_new);
}

__global__ void gradient_displacement_kernel(const float *vertex0, const float *vertex1, int nVertex, const float *pole, const double *Y, const double *coeff, int degree, int deg_beg, int deg_end, double normalization, const float *u1, const float *u2, double *gradient, double *gradient_raw, double *gradient_diag, double *dEdx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < nVertex)
	{
		cuCoordinate coord;
		int nCoeff = (degree + 1) * (degree + 1);
		int n = (deg_end + 1) * (deg_end + 1);
		int n0 = deg_beg * deg_beg;
		int size = n - n0;

		const float *x_bar = &vertex0[i * 3];
		double delta[3] = {0, 0, 0};
		for (int j = 0; j < nCoeff; j++)
		{
			delta[0] += Y[nCoeff * i + j] * coeff[j];
			delta[1] += Y[nCoeff * i + j] * coeff[nCoeff + j];
			delta[2] += Y[nCoeff * i + j] * coeff[nCoeff * 2 + j];
		}
		
		// z
		float z_hat[3];
		coord.sph2cart(pole[0], pole[1], z_hat);
		float z_dot_orth[3]; const float *z_dot_orth_ = (cuVector(u1) * (float)delta[1] + cuVector(u2) * (float)delta[2]).unit().fv();
		for (int j = 0; j < 3; j++) z_dot_orth[j] = z_dot_orth_[j];
		float degree = (float)sqrt(delta[1] * delta[1] + delta[2] * delta[2]);
		cuVector Z_ddot = cuVector(z_hat).cross(cuVector(z_dot_orth));
		float z_ddot[3]; const float *z_ddot_ = Z_ddot.unit().fv();
		for (int j = 0; j < 3; j++) z_ddot[j] = z_ddot_[j];
		float rot[9];
		coord.rotation(z_ddot, degree, rot);
		float z_dot[3];
		coord.rotPoint(z_hat, rot, z_dot);

		// x_hat
		const float *x_hat = &vertex1[i * 3];
		
		cuVector X_hat(x_hat), X_bar(x_bar);
		double x_hat_dot_x_bar = X_hat * X_bar;
		
		//cout << "x_hat_dot_x_bar: " << x_hat_dot_x_bar << endl;
		double dxdg = 0, dxdu2 = 0, dxdu1 = 0;
		dEdx[i] = 0;
		gradient_diag[i] = 0;
		if (fabs(x_hat_dot_x_bar) < 0.999999)	// prevent too much divergence: 1 = x_hat and x_bar matched
		{
			// z_hatXu
			/*const float *z_hatXu1 = Vector(z_hat).cross(Vector(u1)).fv();
			const float *z_hatXu2 = Vector(z_hat).cross(Vector(u2)).fv();*/
			float z_hatXu1[3]; const float *z_hatXu1_ = cuVector(z_hat).cross(cuVector(u1)).fv();
			for (int j = 0; j < 3; j++) z_hatXu1[j] = z_hatXu1_[j];
			float z_hatXu2[3]; const float *z_hatXu2_ = cuVector(z_hat).cross(cuVector(u2)).fv();
			for (int j = 0; j < 3; j++) z_hatXu2[j] = z_hatXu2_[j];
		
			// [z_dot]x
			double z_dot_x[3][3] = {{0, -z_dot[2], z_dot[1]},
								 {z_dot[2], 0, -z_dot[0]},
								 {-z_dot[1], z_dot[0], 0}};
			// [z_hatXu1]x
			double z_hatXu1_x[3][3] = {{0, -z_hatXu1[2], z_hatXu1[1]},
									 {z_hatXu1[2], 0, -z_hatXu1[0]},
									 {-z_hatXu1[1], z_hatXu1[0], 0}};
			// [z_hatXu2]x
			double z_hatXu2_x[3][3] = {{0, -z_hatXu2[2], z_hatXu2[1]},
									 {z_hatXu2[2], 0, -z_hatXu2[0]},
									 {-z_hatXu2[1], z_hatXu2[0], 0}};

			// dx_hat/dgamma = (z_dot_x * x_hat) * x_bar
			dxdg = (z_dot_x[0][0] * x_hat[0] + z_dot_x[0][1] * x_hat[1] + z_dot_x[0][2] * x_hat[2]) * x_bar[0] +
					(z_dot_x[1][0] * x_hat[0] + z_dot_x[1][1] * x_hat[1] + z_dot_x[1][2] * x_hat[2]) * x_bar[1] +
					(z_dot_x[2][0] * x_hat[0] + z_dot_x[2][1] * x_hat[1] + z_dot_x[2][2] * x_hat[2]) * x_bar[2];
		
			// dx_hat/du1 = (z_hatXu1_x * x_hat) * x_bar
			dxdu1 = (z_hatXu1_x[0][0] * x_hat[0] + z_hatXu1_x[0][1] * x_hat[1] + z_hatXu1_x[0][2] * x_hat[2]) * x_bar[0] +
					(z_hatXu1_x[1][0] * x_hat[0] + z_hatXu1_x[1][1] * x_hat[1] + z_hatXu1_x[1][2] * x_hat[2]) * x_bar[1] +
					(z_hatXu1_x[2][0] * x_hat[0] + z_hatXu1_x[2][1] * x_hat[1] + z_hatXu1_x[2][2] * x_hat[2]) * x_bar[2];
		
			// dx_hat/du2 = (z_hatXu1_x * x_hat) * x_bar
			dxdu2 = (z_hatXu2_x[0][0] * x_hat[0] + z_hatXu2_x[0][1] * x_hat[1] + z_hatXu2_x[0][2] * x_hat[2]) * x_bar[0] +
					(z_hatXu2_x[1][0] * x_hat[0] + z_hatXu2_x[1][1] * x_hat[1] + z_hatXu2_x[1][2] * x_hat[2]) * x_bar[1] +
					(z_hatXu2_x[2][0] * x_hat[0] + z_hatXu2_x[2][1] * x_hat[1] + z_hatXu2_x[2][2] * x_hat[2]) * x_bar[2];
					
			double one_minus_x_hat_dot_x_bar_sq = 1.0 - x_hat_dot_x_bar * x_hat_dot_x_bar;
			
			// dE/dx
			dEdx[i] = 2 * acos(x_hat_dot_x_bar);
			double drdx = -1.0 / sqrt(one_minus_x_hat_dot_x_bar_sq);
			dxdg *= drdx;
			dxdu1 *= drdx;
			dxdu2 *= drdx;
			gradient_diag[i] = normalization;
		}
		
		for (int j = n0; j < n; j++)
		{
			/*atomicAdd(&gradient[j], Y[nCoeff * i + j] * dxdg * dEdx[i]);
			atomicAdd(&gradient[nCoeff + j], Y[nCoeff * i + j] * dxdu1 * dEdx[i]);
			atomicAdd(&gradient[nCoeff * 2 *  + j], Y[nCoeff * i + j] * dxdu2 * dEdx[i]);*/
			gradient_raw[size * 3 * i + j - n0] = Y[nCoeff * i + j] * dxdg;
			gradient_raw[size * 3 * i + size + j - n0] = Y[nCoeff * i + j] * dxdu1;
			gradient_raw[size * 3 * i + size * 2 + j - n0] = Y[nCoeff * i + j] * dxdu2;
		}
	}
}

void Gradient::updateGradientDsiplacement(const float *vertex0, const float *vertex1, int nVertex, const float *pole, const double *Y, const double *coeff, int degree, int deg_beg, int deg_end, double normalization, const float *u1, const float *u2, double *gradient, double *M, bool hessian)
{
	int blocksize = 256; // or any size up to 512
	int nblocks = (nVertex + blocksize - 1) / blocksize;
	
	float *d_vertex0;
	float *d_vertex1;
	float *d_pole;
	double *d_Y;
	double *d_coeff;
	float *d_u1;
	float *d_u2;
	double *d_gradient;
	double *d_gradient_new;
	double *d_gradient_raw;
	double *d_gradient_diag;
	double *d_dEdx;
	double *d_M;
	double *d_M_new;
	int n = (deg_end + 1) * (deg_end + 1);
	int n0 = deg_beg * deg_beg;
	int size = n - n0;
	int nblocks2 = (size * 3 + blocksize - 1) / blocksize;
	
	hipMalloc(&d_vertex0, nVertex * 3 * sizeof(float));
	hipMalloc(&d_vertex1, nVertex * 3 * sizeof(float));
	hipMalloc(&d_pole, 2 * sizeof(float));
	hipMalloc(&d_Y, nVertex * (degree + 1) * (degree + 1) * sizeof(double));
	hipMalloc(&d_coeff, (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_u1, 3 * sizeof(float));
	hipMalloc(&d_u2, 3 * sizeof(float));
	hipMalloc(&d_gradient, (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_gradient_new, (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_gradient_raw, nVertex * (degree + 1) * (degree + 1) * 3 * sizeof(double));
	hipMalloc(&d_gradient_diag, nVertex * sizeof(double));
	hipMalloc(&d_dEdx, nVertex * sizeof(double));
	hipMalloc(&d_M, size * 3 * size * 3 * sizeof(double));
	hipMalloc(&d_M_new, size * 3 * size * 3 * sizeof(double));
	
	hipMemcpy(d_vertex0, vertex0, nVertex * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vertex1, vertex1, nVertex * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_pole, pole, 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, nVertex * (degree + 1) * (degree + 1) * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_coeff, coeff, (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_u1, u1, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u2, u2, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gradient, gradient, (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_M, M, size * 3 * size * 3 * sizeof(double), hipMemcpyHostToDevice);

	gradient_displacement_kernel<<<nblocks,blocksize>>>(d_vertex0, d_vertex1, nVertex, d_pole, d_Y, d_coeff, degree, deg_beg, deg_end, normalization, d_u1, d_u2, d_gradient_new, d_gradient_raw, d_gradient_diag, d_dEdx);
	dEdx_kernel<<<nblocks2,blocksize>>>(nVertex, degree, deg_beg, deg_end, d_gradient_new, d_gradient_raw, d_dEdx);
	_SA(normalization, d_gradient_new, (degree + 1) * (degree + 1) * 3, 1);
	_PA(d_gradient, d_gradient_new, (degree + 1) * (degree + 1) * 3, 1);
	hipMemcpy(gradient, d_gradient, (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyDeviceToHost);

	if (hessian)
	{
		_ATDA(d_gradient_raw, d_gradient_diag, nVertex, size * 3, d_M_new);
		_PA(d_M, d_M_new, size * 3, size * 3);
		hipMemcpy(M, d_M, size * 3 * size * 3 * sizeof(double), hipMemcpyDeviceToHost);
	}
	//hipMemcpy(gradient_raw, d_gradient_raw, nVertex * (degree + 1) * (degree + 1) * 3 * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(gradient_diag, d_gradient_diag, nVertex * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_vertex0);
	hipFree(d_vertex1);
	hipFree(d_pole);
	hipFree(d_Y);
	hipFree(d_coeff);
	hipFree(d_u1);
	hipFree(d_u2);
	hipFree(d_gradient);
	hipFree(d_gradient_new);
	hipFree(d_gradient_raw);
	hipFree(d_gradient_diag);
	hipFree(d_dEdx);
	hipFree(d_M);
	hipFree(d_M_new);
}
